#include "hip/hip_runtime.h"
#include <iostream>
#include <stdgpu/unordered_map.cuh>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

struct A
{
    int a;
    int b;

    __host__ __device__ bool operator==(const A& other) const
    {
        return a == other.a && b == other.b;
    }
};

struct B
{
    int a;
    int b;
};

struct HashA
{
    __host__ __device__ std::size_t operator()(const A& key) const
    {
        return key.a;
    }
};

__global__ void insetr(stdgpu::unordered_map<A, B, HashA> map)
{
    int g_id = blockIdx.x * blockDim.x + threadIdx.x;
    map.emplace(A{g_id, 2 }, B{ 3, 4 });
}

int main() 
{
    stdgpu::unordered_map<A, B, HashA> map = stdgpu::unordered_map<A, B, HashA>::createDeviceObject(16 * 16);

    insetr << <16, 16 >> > (map);
    hipDeviceSynchronize();

    auto range_map = map.device_range();

#define OLD_WORKING_CODE 1
#if OLD_WORKING_CODE
    thrust::host_vector<stdgpu::pair<A, B>> host_pairs(map.size());
    thrust::copy(range_map.begin(), range_map.end(), host_pairs.begin());
#else
    thrust::device_vector<stdgpu::pair<A, B>> device_pairs(map.size());
    thrust::copy(range_map.begin(), range_map.end(), device_pairs.begin());
    thrust::host_vector<stdgpu::pair<A, B>> host_pairs = device_pairs;
#endif
    hipDeviceSynchronize();
    for (const auto& pair : host_pairs)
    {
        std::cout << pair.first.a << " " << pair.first.b << " " << pair.second.a << " " << pair.second.b << std::endl;
    }

    std::cout << "Hello, World!" << std::endl;
    return 0;
}